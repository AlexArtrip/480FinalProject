#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "linearprobing.h"
namespace LinearProbing {
    //! Makes an 64-bit Entry out of a key-value pair for the hash table.
    inline __device__ __host__ KeyValue make_entry(unsigned key, unsigned value) {
        return (KeyValue(key) << 32) + value;
    }

    //! Returns the key of an Entry.
    inline __device__ __host__ unsigned get_key(KeyValue entry) {
        return (unsigned)(entry >> 32);
    }

    //! Returns the value of an Entry.
    inline __device__ __host__ unsigned get_value(KeyValue entry) {
        return (unsigned)(entry & 0xffffffff);
    }
    // 32 bit Murmur3 hash
    __device__ uint hash(uint k, uint capacity)
    {
        k ^= k >> 16;
        k *= 0x85ebca6b;
        k ^= k >> 13;
        k *= 0xc2b2ae35;
        k ^= k >> 16;
        return k & (capacity - 1);
    }

    // Create a hash table. For linear probing, this is just an array of KeyValues
    KeyValue* create_hashtable(uint capacity)
    {
        // Allocate memory
        KeyValue* hashtable;
        hipMalloc(&hashtable, sizeof(KeyValue) * capacity);

        // Initialize hash table to empty
        static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
        hipMemset(hashtable, 0xff, sizeof(KeyValue) * capacity);

        return hashtable;
    }

    // Insert the key/values in kvs into the hashtable
    __global__ void gpu_hashtable_insert(KeyValue* hashtable, uint capacity, const KeyValue* kvs, unsigned int numkvs)
    {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs)
        {
            uint key = get_key(kvs[threadid]);
            uint value = get_value(kvs[threadid]);
            uint slot = hash(key, capacity);

            while (true)
            {
                uint prev = atomicCAS((uint *)&hashtable[slot], kEmpty, key);
                if (prev == kEmpty || prev == key)
                {
                    hashtable[slot] = kvs[threadid];
                    return;
                }

                slot = (slot + 1) & (capacity - 1);
            }
        }
    }

    void insert_hashtable(KeyValue* pHashTable, uint capacity, const KeyValue* kvs, uint num_kvs)
    {
        // Copy the keyvalues to the GPU
        KeyValue* device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint)num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_insert << <gridsize, threadblocksize >> > (pHashTable, capacity, device_kvs, (uint)num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU inserted %d items in %f ms (%f million keys/second)\n",
            num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Lookup keys in the hashtable, and return the values
    __global__ void gpu_hashtable_lookup(KeyValue* hashtable, uint capacity, KeyValue* kvs, unsigned int numkvs)
    {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs)
        {
            uint key = get_key(kvs[threadid]);
            uint slot = hash(key, capacity);

            while (true)
            {
                if (get_key(hashtable[slot]) == key)
                {
                    kvs[threadid] = hashtable[slot];
                    return;
                }
                if (get_key(hashtable[slot]) == kEmpty)
                {
                    kvs[threadid] = make_entry(key, kEmpty);
                    return;
                }
                slot = (slot + 1) & (capacity - 1);
            }
        }
    }

    void lookup_hashtable(KeyValue* pHashTable, uint capacity, KeyValue* kvs, uint num_kvs)
    {
        // Copy the keyvalues to the GPU
        KeyValue* device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint)num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_lookup << <gridsize, threadblocksize >> > (pHashTable, capacity, device_kvs, (uint)num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU lookup %d items in %f ms (%f million keys/second)\n",
            num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Delete each key in kvs from the hash table, if the key exists
    // A deleted key is left in the hash table, but its value is set to kEmpty
    // Deleted keys are not reused; once a key is assigned a slot, it never moves
    __global__ void gpu_hashtable_delete(KeyValue* hashtable, uint capacity, const KeyValue* kvs, unsigned int numkvs)
    {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs)
        {
            uint key = get_key(kvs[threadid]);
            uint slot = hash(key, capacity);
            while (true)
            {
                if (get_key(hashtable[slot]) == key)
                {
                    hashtable[slot] = make_entry(key, kEmpty);
                    return;
                }
                if (get_key(hashtable[slot]) == kEmpty)
                {
                    return;
                }
                slot = (slot + 1) & (capacity - 1);
            }
        }
    }

    void delete_hashtable(KeyValue* pHashTable, uint capacity, const KeyValue* kvs, uint num_kvs)
    {
        // Copy the keyvalues to the GPU
        KeyValue* device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint)num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_delete << <gridsize, threadblocksize >> > (pHashTable, capacity, device_kvs, (uint)num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
            num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Iterate over every item in the hashtable; return non-empty key/values
    __global__ void gpu_iterate_hashtable(KeyValue* pHashTable, uint capacity, KeyValue* kvs, uint* kvs_size)
    {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < capacity)
        {
            if (get_key(pHashTable[threadid]) != kEmpty)
            {
                uint value = get_value(pHashTable[threadid]);
                if (value != kEmpty)
                {
                    uint size = atomicAdd(kvs_size, 1);
                    kvs[size] = pHashTable[threadid];
                }
            }
        }
    }

    std::vector<KeyValue> iterate_hashtable(KeyValue* pHashTable, uint capacity)
    {
        uint* device_num_kvs;
        hipMalloc(&device_num_kvs, sizeof(uint));
        hipMemset(device_num_kvs, 0, sizeof(uint));

        KeyValue* device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

        int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
        gpu_iterate_hashtable << <gridsize, threadblocksize >> > (pHashTable, capacity, device_kvs, device_num_kvs);

        uint num_kvs;
        hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint), hipMemcpyDeviceToHost);

        std::vector<KeyValue> kvs;
        kvs.resize(num_kvs);

        hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

        hipFree(device_kvs);
        hipFree(device_num_kvs);

        return kvs;
    }

    // Free the memory of the hashtable
    void destroy_hashtable(KeyValue* pHashTable)
    {
        hipFree(pHashTable);
    }
}