#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "cuckoo.h"

namespace Cuckoo {
    //! Makes an 64-bit Entry out of a key-value pair for the hash table.
    inline __device__ __host__ KeyValue make_entry(unsigned key, unsigned value) {
        return (KeyValue(key) << 32) + value;
    }

    //! Returns the key of an Entry.
    inline __device__ __host__ unsigned get_key(KeyValue entry) {
        return (unsigned)(entry >> 32);
    }

    //! Returns the value of an Entry.
    inline __device__ __host__ unsigned get_value(KeyValue entry) {
        return (unsigned)(entry & 0xffffffff);
    }


    inline __device__ __host__ unsigned stash_hash_function(const unsigned key) {   //TODO:: might need to change type sig
        return (2720648079 ^ key + 13) % kStashSize;
    }
    // 32 bit Murmur3 hash
    __device__ uint hash(int hash_id, uint k, uint capacity) {
        k ^= k >> 16;
        if (hash_id == 0) {
            k *= 0x85ebca6b;
            k ^= k >> 13;
            k *= 0xc2b2ae35;
        } else {
            k *= 0xcc9e2d51;
            k ^= k >> 13;
            k *= 0x1b873593;
        }
        k ^= k >> 16;
        return k & (capacity - 1);

    }

    //! Determine where to insert the key next.  The hash functions are used in round-robin order.
    __device__ unsigned determine_next_location(const unsigned table_size,
                                                const unsigned key,
                                                const unsigned previous_location) {
        uint next_location = hash(0, key, table_size);
        if (next_location == previous_location) {
            return hash(1, key, table_size);
        }
        return next_location;
    }

    // Create a hash table. For linear probing, this is just an array of KeyValues
    KeyValue *create_hashtable(uint capacity) {
        // Allocate memory
        KeyValue *hashtable;
        hipMalloc(&hashtable, sizeof(KeyValue) * (capacity + kStashSize));

        // Initialize hash table to empty
        static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
        hipMemset(hashtable, 0xff, sizeof(KeyValue) * (capacity + kStashSize));

        return hashtable;
    }

    // Insert the key/values in kvs into the hashtable
    __global__ void gpu_hashtable_insert(KeyValue *hashtable, uint capacity, uint max_iteration_attempts,
                                         const KeyValue *kvs, unsigned int numkvs) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            KeyValue entry = kvs[threadid];
            unsigned key = get_key(entry);
            // The key is always inserted into its first slot at the start.
            uint location = hash(0, key, capacity);

            // Keep inserting until an empty slot is found or the eviction chain grows too large.
            for (unsigned its = 1; its <= max_iteration_attempts; its++) {
                // Insert the new entry.
                entry = atomicExch(&hashtable[location], entry);
                key = get_key(entry);
                // If no key was evicted, we're done.
                if (key == kEmpty) {
                    // *iterations_used = its;
                    break;
                }

                // Otherwise, determine where the evicted key will go.
                location = determine_next_location(capacity, key, location);
            }

            if (key != kEmpty) {
                // Shove it into the stash.
                unsigned slot = stash_hash_function(key);
                KeyValue *stash = hashtable + capacity;
                KeyValue replaced_entry = atomicCAS((stash + slot), kvEmpty, entry);
//                if (replaced_entry != kEmpty) {
//                    return false;
//                } else {
//                    atomicAdd(stash_count, 1);
//                }
            }
        }
    }

    void insert_hashtable(KeyValue *pHashTable, uint capacity, uint max_iteration_attempts, const KeyValue *kvs,
                          uint num_kvs) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

//        unsigned *d_stash_count = NULL;
//        CUDA_SAFE_CALL(hipMalloc((void**)&d_stash_count, sizeof(unsigned)));

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_insert <<<gridsize, threadblocksize>>>(pHashTable, capacity, max_iteration_attempts,
                                                             device_kvs, (uint) num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU inserted %d items in %f ms (%f million keys/second)\n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Lookup keys in the hashtable, and return the values
    __global__ void gpu_hashtable_lookup(KeyValue *hashtable, uint capacity, KeyValue *kvs, unsigned int numkvs) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            uint key = get_key(kvs[threadid]);
            KeyValue slot0 = hashtable[hash(0, key, capacity)];
            if (get_key(slot0) == key) {
                kvs[threadid] = slot0;
                return;
            }
            KeyValue slot1 = hashtable[hash(1, key, capacity)];
            if (get_key(slot1) == key) {
                kvs[threadid] = slot1;
                return;
            }
            KeyValue stash = hashtable[hash(1, key, capacity)];
            if (get_key(stash) == key) {
                kvs[threadid] = stash;
                return;
            }
            kvs[threadid] = make_entry(key, kEmpty);
        }
    }

    void lookup_hashtable(KeyValue *pHashTable, uint capacity, KeyValue *kvs, uint num_kvs) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_lookup <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, (uint) num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU lookup %d items in %f ms (%f million keys/second)\n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Delete each key in kvs from the hash table, if the key exists
    // A deleted key is left in the hash table, but its value is set to kEmpty
    // Deleted keys are not reused; once a key is assigned a slot, it never moves
    __global__ void gpu_hashtable_delete(KeyValue *hashtable, uint capacity, const KeyValue *kvs, unsigned int numkvs) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            uint key = get_key(kvs[threadid]);
            // TODO fix!!!
            KeyValue slot0 = hashtable[hash(0, key, capacity)];
            if (get_key(slot0) == key) {
                hashtable[threadid] = kvEmpty;
                return;
            }
            KeyValue slot1 = hashtable[hash(1, key, capacity)];
            if (get_key(slot1) == key) {
                hashtable[threadid] = kvEmpty;
                return;
            }
            KeyValue stash = hashtable[hash(1, key, capacity)];
            if (get_key(stash) == key) {
                hashtable[threadid] = kvEmpty;
                return;
            }
        }
    }

    void delete_hashtable(KeyValue *pHashTable, uint capacity, const KeyValue *kvs, uint num_kvs) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_delete <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, (uint) num_kvs);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Iterate over every item in the hashtable; return non-empty key/values
    __global__ void gpu_iterate_hashtable(KeyValue *pHashTable, uint capacity, KeyValue *kvs, uint *kvs_size) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < capacity) {
            if (get_key(pHashTable[threadid]) != kEmpty) {
                uint value = get_value(pHashTable[threadid]);
                if (value != kEmpty) {
                    uint size = atomicAdd(kvs_size, 1);
                    kvs[size] = pHashTable[threadid];
                }
            }
        }
    }

    std::vector <KeyValue> iterate_hashtable(KeyValue *pHashTable, uint capacity) {
        uint *device_num_kvs;
        hipMalloc(&device_num_kvs, sizeof(uint));
        hipMemset(device_num_kvs, 0, sizeof(uint));

        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

        int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
        gpu_iterate_hashtable <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, device_num_kvs);

        uint num_kvs;
        hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint), hipMemcpyDeviceToHost);

        std::vector <KeyValue> kvs;
        kvs.resize(num_kvs);

        hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

        hipFree(device_kvs);
        hipFree(device_num_kvs);

        return kvs;
    }

    // Free the memory of the hashtable
    void destroy_hashtable(KeyValue *pHashTable) {
        hipFree(pHashTable);
    }
}