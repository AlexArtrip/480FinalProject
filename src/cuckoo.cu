#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "cuckoo.h"
#include "cuda_util.h"
#include <chrono>
#include <thread>

namespace Cuckoo {
    unsigned ComputeMaxIterations(const unsigned n,
        const unsigned table_size,
        const unsigned num_functions) {
        float lg_input_size = (float)(log((double)n) / log(2.0));

        // #define CONSTANT_ITERATIONS
        //#ifdef CONSTANT_ITERATIONS
        //        // Set the maximum number of iterations to 7lg(N).
        //    const unsigned MAX_ITERATION_CONSTANT = 7;
        //    unsigned max_iterations = MAX_ITERATION_CONSTANT * lg_input_size;
        //#else
                // Use an empirical formula for determining what the maximum number of
                // iterations should be.  Works OK in most situations.
        float load_factor = float(n) / table_size;
        float ln_load_factor = (float)(log(load_factor) / log(2.71828183));

        unsigned max_iterations = (unsigned)(4.0 * ceil(-1.0 / (0.028255 + 1.1594772 *
            ln_load_factor) * lg_input_size));
        //#endif
        return max_iterations;
    }

    //! Makes an 64-bit Entry out of a key-value pair for the hash table.
    inline __device__ __host__ KeyValue make_entry(unsigned key, unsigned value) {
        return (KeyValue(key) << 32) + value;
    }

    //! Returns the key of an Entry.
    inline __device__ __host__ unsigned get_key(KeyValue entry) {
        return (unsigned)(entry >> 32);
    }

    //! Returns the value of an Entry.
    inline __device__ __host__ unsigned get_value(KeyValue entry) {
        return (unsigned)(entry & 0xffffffff);
    }


    inline __device__ __host__ unsigned stash_hash_function(const unsigned key) {   //TODO:: might need to change type sig
        return (2720648079 ^ key + 13) % kStashSize;
    }
    // 32 bit Murmur3 hash
    __device__ uint hash(int hash_id, uint k, uint capacity) {
        k ^= k >> 16;
        if (hash_id == 0) {
            k *= 0x85ebca6b;
            k ^= k >> 13;
            k *= 0xc2b2ae35;
        } else {
            k *= 0xcc9e2d51;
            k ^= k >> 13;
            k *= 0x1b873593;
        }
        k ^= k >> 16;
        return k & (capacity - 1);

    }

    //! Determine where to insert the key next.  The hash functions are used in round-robin order.
    __device__ unsigned determine_next_location(const unsigned table_size,
                                                const unsigned key,
                                                const unsigned previous_location) {
        uint next_location = hash(0, key, table_size);
        if (next_location == previous_location) {
            //return next_location + 1;
            return hash(1, key, table_size);
        }
        return next_location;
    }

    // Create a hash table. For linear probing, this is just an array of KeyValues
    KeyValue *create_hashtable(uint capacity, uint** stash_count) {
        // Allocate memory
        KeyValue *hashtable;
        hipMalloc(&hashtable, sizeof(KeyValue) * (capacity + kStashSize));

        // Initialize hash table to empty
        static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
        hipMemset(hashtable, 0xff, sizeof(KeyValue) * (capacity + kStashSize));

        CUDA_SAFE_CALL(hipMalloc((void**)stash_count, sizeof(uint)));
        CUDA_SAFE_CALL(hipMemset(*stash_count, 0, sizeof(uint)));

        //printf("Hash table created successfully");
        //std::chrono::seconds dura(5);
        //std::this_thread::sleep_for(dura);
        return hashtable;
    }

    // Insert the key/values in kvs into the hashtable
    __global__ void gpu_hashtable_insert(KeyValue *hashtable, uint capacity, uint max_iteration_attempts,
                                         const KeyValue *kvs, unsigned int numkvs,
                                         uint *stash_count, uint *fail_count) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            KeyValue entry = kvs[threadid];
            unsigned key = get_key(entry);
            unsigned prev_key = key;
            // The key is always inserted into its first slot at the start.
            uint location = hash(0, key, capacity);

            // Keep inserting until an empty slot is found or the eviction chain grows too large.
            for (unsigned its = 1; its <= max_iteration_attempts; its++) {
                // Insert the new entry.
                prev_key = key;
                entry = atomicExch(&hashtable[location], entry);
                key = get_key(entry);
                // If no key was evicted or this key is already present, we're done.
                if (key == kEmpty || prev_key == key) {
                    // *iterations_used = its;
                    return;
                }
                // Otherwise, determine where the evicted key will go.
                location = determine_next_location(capacity, key, location);
            }

            if (key != kEmpty) {
                //printf("failed insert will stash now after max_iter = %u \n", max_iteration_attempts);
                // Shove it into the stash.
                uint slot = stash_hash_function(key);
                KeyValue *stash = hashtable + capacity;
                KeyValue replaced_entry = atomicCAS((stash + slot), kvEmpty, entry);
                if (replaced_entry != kvEmpty) {
                    atomicAdd(fail_count, 1);
                } else {
                    atomicAdd(stash_count, 1);
                }
            }
        }
    }

    void insert_hashtable(KeyValue *pHashTable, uint capacity, uint max_iteration_attempts, const KeyValue *kvs,
                          uint num_kvs, uint *d_stash_count) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

//        unsigned* d_stash_count = NULL;
//        CUDA_SAFE_CALL(hipMalloc((void**)&d_stash_count, sizeof(uint)));
//        CUDA_SAFE_CALL(hipMemset(d_stash_count, 0, sizeof(uint)));
        unsigned* d_fail_count = NULL;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_fail_count, sizeof(uint)));
        CUDA_SAFE_CALL(hipMemset(d_fail_count, 0, sizeof(uint)));

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_insert <<<gridsize, threadblocksize>>>(pHashTable, capacity, max_iteration_attempts,
                                                             device_kvs, (uint) num_kvs,
                                                             d_stash_count, d_fail_count);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;

        printf("    GPU inserted %d items in %f ms (%f million keys/second) \n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        // Copy out the stash size.
        uint stash_count;
        CUDA_SAFE_CALL(hipMemcpy(&stash_count, d_stash_count, sizeof(unsigned), hipMemcpyDeviceToHost));
        if (stash_count != 0) {
            printf("        stash count is %u\n", stash_count);
        }
        // Copy out the stash size.
        uint fail_count;
        CUDA_SAFE_CALL(hipMemcpy(&fail_count, d_fail_count, sizeof(unsigned), hipMemcpyDeviceToHost));
        if (fail_count != 0) {
            printf("        fail count is %u\n", fail_count);
        }
        hipFree(d_fail_count);

        hipFree(device_kvs);
    }

    // Lookup keys in the hashtable, and return the values
    __global__ void gpu_hashtable_lookup(KeyValue *hashtable, uint capacity, KeyValue *kvs,
                                         unsigned int numkvs, uint* stash_count) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            uint key = get_key(kvs[threadid]);
            uint hash_val = hash(0, key, capacity);
            KeyValue slot0 = hashtable[hash_val];
            if (get_key(slot0) == key) {
                kvs[threadid] = slot0;
                return;
            }
            KeyValue slot1 = hashtable[hash(1, key, capacity)];
            //KeyValue slot1 = hashtable[hash_val + 1];
            if (get_key(slot1) == key) {
                kvs[threadid] = slot1;
                return;
            }
            if (*stash_count) {
                uint slot = stash_hash_function(key);
                KeyValue *stash = hashtable + capacity;
                KeyValue entry = stash[slot];
                if (get_key(entry) == key) {
                    kvs[threadid] = entry;
                    return;
                }
            }
            kvs[threadid] = make_entry(key, kEmpty);
        }
    }

    void lookup_hashtable(KeyValue *pHashTable, uint capacity, KeyValue *kvs, uint num_kvs, uint* stash_count) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_lookup <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, (uint) num_kvs,
                                                                stash_count);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU lookup %d items in %f ms (%f million keys/second)\n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Delete each key in kvs from the hash table, if the key exists
    // A deleted key is left in the hash table, but its value is set to kEmpty
    // Deleted keys are not reused; once a key is assigned a slot, it never moves
    __global__ void gpu_hashtable_delete(KeyValue *hashtable, uint capacity, const KeyValue *kvs,
                                         unsigned int numkvs, uint* stash_count) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < numkvs) {
            uint key = get_key(kvs[threadid]);
            // TODO fix!!!
            KeyValue slot0 = hashtable[hash(0, key, capacity)];
            if (get_key(slot0) == key) {
                hashtable[threadid] = kvEmpty;
                return;
            }
            KeyValue slot1 = hashtable[hash(1, key, capacity)];
            if (get_key(slot1) == key) {
                hashtable[threadid] = kvEmpty;
                return;
            }
            if (*stash_count) {
                uint slot = stash_hash_function(key);
                KeyValue *stash = hashtable + capacity;
                KeyValue entry = stash[slot];
                if (get_key(entry) == key) {
                    stash[slot] = kvEmpty;
                    return;
                }
            }
        }
    }

    void delete_hashtable(KeyValue *pHashTable, uint capacity, const KeyValue *kvs, uint num_kvs, uint* stash_count) {
        // Copy the keyvalues to the GPU
        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
        hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

        // Have CUDA calculate the thread block size
        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

        // Create events for GPU timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        // Insert all the keys into the hash table
        int gridsize = ((uint) num_kvs + threadblocksize - 1) / threadblocksize;
        gpu_hashtable_delete <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, (uint) num_kvs,
                                                                stash_count);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        float seconds = milliseconds / 1000.0f;
        printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
               num_kvs, milliseconds, num_kvs / (double) seconds / 1000000.0f);

        hipFree(device_kvs);
    }

    // Iterate over every item in the hashtable; return non-empty key/values
    __global__ void gpu_iterate_hashtable(KeyValue *pHashTable, uint capacity, KeyValue *kvs, uint *kvs_size) {
        unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadid < capacity) {
            if (get_key(pHashTable[threadid]) != kEmpty) {
                uint value = get_value(pHashTable[threadid]);
                if (value != kEmpty) {
                    uint size = atomicAdd(kvs_size, 1);
                    kvs[size] = pHashTable[threadid];
                }
            }
        }
    }

    std::vector <KeyValue> iterate_hashtable(KeyValue *pHashTable, uint capacity) {
        uint *device_num_kvs;
        hipMalloc(&device_num_kvs, sizeof(uint));
        hipMemset(device_num_kvs, 0, sizeof(uint));

        KeyValue *device_kvs;
        hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

        int mingridsize;
        int threadblocksize;
        hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

        int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
        gpu_iterate_hashtable <<< gridsize, threadblocksize >>> (pHashTable, capacity, device_kvs, device_num_kvs);

        uint num_kvs;
        hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint), hipMemcpyDeviceToHost);

        std::vector <KeyValue> kvs;
        kvs.resize(num_kvs);

        hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

        hipFree(device_kvs);
        hipFree(device_num_kvs);

        return kvs;
    }

    // Free the memory of the hashtable
    void destroy_hashtable(KeyValue *pHashTable) {
        hipFree(pHashTable);
    }
}